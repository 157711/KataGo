#include "hip/hip_runtime.h"

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include <hip/hip_fp16.h>
#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/iterator/discard_iterator.h>

#include <stdexcept>

#include "../neuralnet/cudahelpers.h"

#if __CUDA_ARCH__ >= 530
#define CUDA_SUPPORTS_FP16
#endif


template <typename T>
__global__
void channelConcatKernel(
  const T* inA,
  const T* inB,
  T* out,
  int chwA,
  int chwB,
  int numBlocksA,
  int numBlocksB,
  int n
) {
  if(blockIdx.x < numBlocksA) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < chwA) {
      int nchwA = n*chwA;
      int chwOut = (chwA+chwB);

      int aIdx = index;
      int outIdx = index;
      while(aIdx < nchwA) {
        out[outIdx] = inA[aIdx];
        aIdx += chwA;
        outIdx += chwOut;
      }
    }
  }
  else {
    int index = (blockIdx.x - numBlocksA) * blockDim.x + threadIdx.x;
    if(index < chwB) {
      int nchwB = n*chwB;
      int chwOut = (chwA+chwB);

      int bIdx = index;
      int outIdx = chwA+index;
      while(bIdx < nchwB) {
        out[outIdx] = inB[bIdx];
        bIdx += chwB;
        outIdx += chwOut;
      }
    }
  }
}

template <typename T>
void customCudaChannelConcatTemplate(const T* inA, const T* inB, T* out, int chwA, int chwB, int n) {
  //TODO maybe tune this number, it varies by GPU
  int targetNumThreads = 256;
  int blockSize = targetNumThreads;
  int numBlocksA = (chwA + blockSize-1) / blockSize;
  int numBlocksB = (chwB + blockSize-1) / blockSize;
  int numBlocks = numBlocksA + numBlocksB;
  channelConcatKernel<<<numBlocks, blockSize>>>(inA,inB,out,chwA,chwB,numBlocksA,numBlocksB,n);
}
template void customCudaChannelConcatTemplate<float>(const float* inA, const float* inB, float* out, int chwA, int chwB, int n);
template void customCudaChannelConcatTemplate<half>(const half* inA, const half* inB, half* out, int chwA, int chwB, int n);

void customCudaChannelConcat(const float* inA, const float* inB, float* out, int chwA, int chwB, int n) {
  customCudaChannelConcatTemplate<float>(inA,inB,out,chwA,chwB,n);
}
void customCudaChannelConcat(const half* inA, const half* inB, half* out, int chwA, int chwB, int n) {
  customCudaChannelConcatTemplate<half>(inA,inB,out,chwA,chwB,n);
}


template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
  T c;
  __host__ __device__ linear_index_to_row_index(T c) : c(c) {}
  __host__ __device__ T operator()(T i) { return i / c; }
};

void customCudaPoolRowsSum(float* in, float* out, int n, int c) {

  thrust::device_ptr<float> inThrust = thrust::device_pointer_cast(in);
  thrust::device_ptr<float> outThrust = thrust::device_pointer_cast(out);

  thrust::reduce_by_key(
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(c)),
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(c)) + (n*c),
    inThrust,
    thrust::make_discard_iterator(),
    outThrust
  );

}

void customCudaPoolRowsMax(float* in, float* out, int n, int c) {

  thrust::device_ptr<float> inThrust = thrust::device_pointer_cast(in);
  thrust::device_ptr<float> outThrust = thrust::device_pointer_cast(out);

  thrust::reduce_by_key(
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(c)),
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(c)) + (n*c),
    inThrust,
    thrust::make_discard_iterator(),
    outThrust,
    thrust::equal_to<int>(),
    thrust::maximum<float>()
  );

}


__global__
void nchwTransposeKernel(const float *in, float* out, int xSize, int ySize, int tileDim, int tileStride, int xySize)
{
  //+1 avoids bank conflicts
  extern __shared__ float tileNCHW[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int nc = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = xIdx + xSize * (yIdx+j) + xySize * nc;
      tileNCHW[(threadIdx.y+j)*tileDimP1 + threadIdx.x] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = outXIdx + ySize * (outYIdx+j) + xySize * nc;
      out[outIdx] = tileNCHW[threadIdx.x*tileDimP1 + threadIdx.y+j];
    }
  }
}

__global__
void nhwcTransposeKernel(const float *in, float* out, int xSize, int ySize, int cSize, int tileDim, int tileStride, int xycSize)
{
  //+1 reduces bank conflicts
  extern __shared__ float tileNHWC[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int cIdx = threadIdx.z;
  int n = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = cIdx + cSize * (xIdx + xSize * (yIdx+j)) + xycSize * n;
      tileNHWC[cIdx + cSize * ((threadIdx.y+j)*tileDimP1 + threadIdx.x)] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = cIdx + cSize * (outXIdx + ySize * (outYIdx+j)) + xycSize * n;
      out[outIdx] = tileNHWC[cIdx + cSize * (threadIdx.x*tileDimP1 + threadIdx.y+j)];
    }
  }
}

__global__
void nchwTransposeHalfKernel(const half *in, half* out, int xSize, int ySize, int tileDim, int tileStride, int xySize)
{
  //+1 avoids bank conflicts
  extern __shared__ half tileNCHWHALF[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int nc = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = xIdx + xSize * (yIdx+j) + xySize * nc;
      tileNCHWHALF[(threadIdx.y+j)*tileDimP1 + threadIdx.x] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = outXIdx + ySize * (outYIdx+j) + xySize * nc;
      out[outIdx] = tileNCHWHALF[threadIdx.x*tileDimP1 + threadIdx.y+j];
    }
  }
}

__global__
void nhwcTransposeHalfKernel(const half *in, half* out, int xSize, int ySize, int cSize, int tileDim, int tileStride, int xycSize)
{
  //+1 reduces bank conflicts
  extern __shared__ half tileNHWCHALF[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int cIdx = threadIdx.z;
  int n = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = cIdx + cSize * (xIdx + xSize * (yIdx+j)) + xycSize * n;
      tileNHWCHALF[cIdx + cSize * ((threadIdx.y+j)*tileDimP1 + threadIdx.x)] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = cIdx + cSize * (outXIdx + ySize * (outYIdx+j)) + xycSize * n;
      out[outIdx] = tileNHWCHALF[cIdx + cSize * (threadIdx.x*tileDimP1 + threadIdx.y+j)];
    }
  }
}

void customCudaNCHWTranspose(const float *in, float* out, int xSize, int ySize, int ncSize) {
  if(ncSize > 65536)
    throw std::runtime_error("customCudaNCHWTranspose: ncSize too large");

  //TODO maybe tune these numbers, it varies by GPU
  //The first one should be the warp size, since it's set to what we need to avoid bank conflicts?
  //Or is it better to just make it xSize, to reduce overhead on top of 19x19?
  int targetNumThreads = 256;
  int tileDim = 32;
  int tileStride = targetNumThreads/tileDim;
  dim3 grid((xSize+tileDim-1)/tileDim,(ySize+tileDim-1)/tileDim,ncSize);
  dim3 threads(tileDim,tileStride,1);
  int sharedMemSize = sizeof(float)*tileDim*(tileDim+1);
  nchwTransposeKernel<<<grid,threads,sharedMemSize>>>(in,out,xSize,ySize,tileDim,tileStride,xSize*ySize);
}

void customCudaNHWCTranspose(const float *in, float* out, int xSize, int ySize, int cSize, int nSize) {
  if(cSize > 64)
    throw std::runtime_error("customCudaNHWCTranspose: cSize too large");
  //TODO maybe tune these numbers, it varies by GPU
  int targetNumThreads = 256;

  int tileDim = 1;
  while(tileDim * 2 * cSize <= targetNumThreads)
    tileDim *= 2;

  int tileStride = 1;
  if(tileDim > 32) {
    tileStride = tileDim / 32;
    tileDim = 32;
  }
  dim3 grid((xSize+tileDim-1)/tileDim,(ySize+tileDim-1)/tileDim,nSize);
  dim3 threads(tileDim,tileStride,cSize);
  int sharedMemSize = sizeof(float)*tileDim*(tileDim+1)*cSize;
  nhwcTransposeKernel<<<grid,threads,sharedMemSize>>>(in,out,xSize,ySize,cSize,tileDim,tileStride,xSize*ySize*cSize);
}

void customCudaNCHWTranspose(const half *in, half* out, int xSize, int ySize, int ncSize) {
  if(ncSize > 65536)
    throw std::runtime_error("customCudaNCHWTranspose: ncSize too large");
  //TODO maybe tune these numbers, it varies by GPU
  //The first one should be the warp size, since it's set to what we need to avoid bank conflicts?
  //Or is it better to just make it xSize, to reduce overhead on top of 19x19?
  int targetNumThreads = 256;
  int tileDim = 32;
  int tileStride = targetNumThreads/tileDim;
  dim3 grid((xSize+tileDim-1)/tileDim,(ySize+tileDim-1)/tileDim,ncSize);
  dim3 threads(tileDim,tileStride,1);
  int sharedMemSize = sizeof(half)*tileDim*(tileDim+1);
  nchwTransposeHalfKernel<<<grid,threads,sharedMemSize>>>(in,out,xSize,ySize,tileDim,tileStride,xSize*ySize);
}

void customCudaNHWCTranspose(const half *in, half* out, int xSize, int ySize, int cSize, int nSize) {
  if(cSize > 64)
    throw std::runtime_error("customCudaNHWCTranspose: cSize too large");
  //TODO maybe tune these numbers, it varies by GPU
  int targetNumThreads = 256;

  int tileDim = 1;
  while(tileDim * 2 * cSize <= targetNumThreads)
    tileDim *= 2;

  int tileStride = 1;
  if(tileDim > 32) {
    tileStride = tileDim / 32;
    tileDim = 32;
  }
  dim3 grid((xSize+tileDim-1)/tileDim,(ySize+tileDim-1)/tileDim,nSize);
  dim3 threads(tileDim,tileStride,cSize);
  int sharedMemSize = sizeof(half)*tileDim*(tileDim+1)*cSize;
  nhwcTransposeHalfKernel<<<grid,threads,sharedMemSize>>>(in,out,xSize,ySize,cSize,tileDim,tileStride,xSize*ySize*cSize);
}

template <typename T>
__global__
void mirrorKernel(const T *in, T* out, int mSize, int subSize)
{
  int subIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int mIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int batchIdx = blockIdx.z;
  if(subIdx < subSize && mIdx < mSize) {
    int inIdx = subIdx + subSize * (mIdx + mSize * batchIdx);
    int outIdx = subIdx + subSize * ((mSize-mIdx-1) + mSize * batchIdx);
    out[outIdx] = in[inIdx];
  }
}

template <typename T>
void customCudaMirrorTemplate(const T *in, T* out, int batchSize, int mSize, int subSize) {
  if(batchSize > 65536)
    throw std::runtime_error("customCudaMirror: batchSize too large");
  if(mSize > 65536)
    throw std::runtime_error("customCudaMirror: mSize too large");

  //TODO maybe tune these numbers, it varies by GPU
  int targetNumThreads = 256;

  int subThreads;
  int subBlocks;
  int mThreads;
  int mBlocks;

  if(subSize > targetNumThreads) {
    subThreads = targetNumThreads/2;
    subBlocks = (subSize + subThreads - 1) / subThreads;
    mThreads = 1;
    mBlocks = mSize;
  }
  else if(subSize > targetNumThreads/2) {
    subThreads = subSize;
    subBlocks = 1;
    mThreads = 1;
    mBlocks = mSize;
  }
  else {
    subThreads = subSize;
    subBlocks = 1;
    mThreads = targetNumThreads / subSize;
    mBlocks = (mSize + mThreads - 1) / mThreads;
  }

  dim3 grid(subBlocks,mBlocks,batchSize);
  dim3 threads(subThreads,mThreads,1);
  mirrorKernel<<<grid,threads>>>(in,out,mSize,subSize);
}

template <typename T>
void customCudaMirrorNCHWTemplate(const T *in, T* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  if(mirrorY && mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*cSize,ySize*xSize,1);
  else if(mirrorY)
    customCudaMirrorTemplate(in,out,batchSize*cSize,ySize,xSize);
  else if(mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*cSize*ySize,xSize,1);
  else
    hipMemcpyAsync(out,in,sizeof(T)*batchSize*cSize*ySize*xSize,hipMemcpyDeviceToDevice);
}

template <typename T>
void customCudaMirrorNHWCTemplate(const T *in, T* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  if(mirrorY && mirrorX)
    customCudaMirrorTemplate(in,out,batchSize,ySize*xSize,cSize);
  else if(mirrorY)
    customCudaMirrorTemplate(in,out,batchSize,ySize,xSize*cSize);
  else if(mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*ySize,xSize,cSize);
  else
    hipMemcpyAsync(out,in,sizeof(T)*batchSize*ySize*xSize*cSize,hipMemcpyDeviceToDevice);
}

void customCudaMirror(const float *in, float* out, int batchSize, int mSize, int subSize) {
  customCudaMirrorTemplate<float>(in,out,batchSize,mSize,subSize);
}
void customCudaMirrorNCHW(const float *in, float* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNCHWTemplate<float>(in,out,batchSize,cSize,ySize,xSize,mirrorY,mirrorX);
}
void customCudaMirrorNHWC(const float *in, float* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNHWCTemplate<float>(in,out,batchSize,ySize,xSize,cSize,mirrorY,mirrorX);
}

void customCudaMirror(const half *in, half* out, int batchSize, int mSize, int subSize) {
  customCudaMirrorTemplate<half>(in,out,batchSize,mSize,subSize);
}
void customCudaMirrorNCHW(const half *in, half* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNCHWTemplate<half>(in,out,batchSize,cSize,ySize,xSize,mirrorY,mirrorX);
}
void customCudaMirrorNHWC(const half *in, half* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNHWCTemplate<half>(in,out,batchSize,ySize,xSize,cSize,mirrorY,mirrorX);
}


__global__
void copyToHalfKernel(const float *in, half* out, int n)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n) {
    out[idx] = __float2half(in[idx]);
  }
}
__global__
void copyFromHalfKernel(const half *in, float* out, int n)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n) {
    out[idx] = __half2float(in[idx]);
  }
}

void customCudaCopyToHalf(const float* in, half* out, int n) {
  //TODO maybe tune these numbers, it varies by GPU
  int targetNumThreads = 256;
  int blockSize = targetNumThreads;
  int numBlocks = (n+blockSize-1)/blockSize;
  copyToHalfKernel<<<numBlocks, blockSize>>>(in,out,n);
}
void customCudaCopyFromHalf(const half* in, float* out, int n) {
  //TODO maybe tune these numbers, it varies by GPU
  int targetNumThreads = 256;
  int blockSize = targetNumThreads;
  int numBlocks = (n+blockSize-1)/blockSize;
  copyFromHalfKernel<<<numBlocks, blockSize>>>(in,out,n);
}

#ifdef CUDA_SUPPORTS_FP16
__global__
void addBiasInplaceHalfKernel(half *buf, const half* biases, int nSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int nIdx = blockIdx.y * blockDim.y + threadIdx.y;
  if(cIdx < cSize && nIdx < nSize) {
    int idx = nIdx * cSize + cIdx;
    buf[idx] = __hadd(buf[idx],biases[cIdx]);
  }
}
#else
__global__
void addBiasInplaceHalfKernel(half *buf, const half* biases, int nSize, int cSize)
{
  //Do nothing, FP16 not supported
}
#endif


void customCudaAddBiasInplace(half* buf, const half* biases, int nSize, int cSize) {
  //TODO maybe tune these numbers, it varies by GPU
  int targetNumThreads = 256;

  int cThreads;
  int cBlocks;
  int nThreads;
  int nBlocks;

  if(cSize > targetNumThreads) {
    cThreads = targetNumThreads/2;
    cBlocks = (cSize + cThreads - 1) / cThreads;
    nThreads = 1;
    nBlocks = nSize;
  }
  else if(cSize > targetNumThreads/2) {
    cThreads = cSize;
    cBlocks = 1;
    nThreads = 1;
    nBlocks = nSize;
  }
  else {
    cThreads = cSize;
    cBlocks = 1;
    nThreads = targetNumThreads / nSize;
    nBlocks = (nSize + nThreads - 1) / nThreads;
  }

  if(nBlocks > 65536)
    throw std::runtime_error("customCudaAddBiasInplace: nSize too large given cSize");

  dim3 grid(cBlocks,nBlocks,1);
  dim3 threads(cThreads,nThreads,1);
  addBiasInplaceHalfKernel<<<grid,threads>>>(buf,biases,nSize,cSize);
}

#ifdef CUDA_SUPPORTS_FP16
__global__
void applyScaleBiasHalfKernel(const half *in, half* out, const half* scale, const half* biases, int cSize, int sSize)
{
  int sIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int cIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int idx = (nIdx * cSize + cIdx) * sSize + sIdx;
    out[idx] = __hfma(in[idx],scale[cIdx],biases[cIdx]);
  }
}
#else
__global__
void applyScaleBiasHalfKernel(const half *in, half* out, const half* scale, const half* biases, int cSize, int sSize)
{
  //Do nothing, FP16 not supported
}
#endif

void customCudaApplyScaleBias(const half* in, half* out, const half* scale, const half* biases, int nSize, int cSize, int sSize) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaApplyScaleBias: nSize too large");
  if(cSize > 65536)
    throw std::runtime_error("customCudaApplyScaleBias: cSize too large");

  //TODO maybe tune these numbers, it varies by GPU
  int targetNumThreads = 256;

  int sThreads;
  int sBlocks;
  int cThreads;
  int cBlocks;

  if(sSize > targetNumThreads) {
    sThreads = targetNumThreads/2;
    sBlocks = (sSize + sThreads - 1) / sThreads;
    cThreads = 1;
    cBlocks = cSize;
  }
  else if(sSize > targetNumThreads/2) {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = 1;
    cBlocks = cSize;
  }
  else {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = targetNumThreads / sSize;
    cBlocks = (cSize + cThreads - 1) / cThreads;
  }

  dim3 grid(sBlocks,cBlocks,nSize);
  dim3 threads(sThreads,cThreads,1);
  applyScaleBiasHalfKernel<<<grid,threads>>>(in,out,scale,biases,cSize,sSize);
}
