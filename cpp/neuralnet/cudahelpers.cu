#include "hip/hip_runtime.h"

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include <hip/hip_fp16.h>
#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/iterator/discard_iterator.h>

#include <stdexcept>

#include "../neuralnet/cudahelpers.h"

#if __CUDA_ARCH__ >= 530
#define CUDA_SUPPORTS_FP16
#endif

//TODO maybe tune this number, it varies by GPU
static const int targetNumThreads = 256;

//--------------------------------------------------------------------------------------------------------------

template <typename T>
__global__
void channelConcatKernel(
  const T* inA,
  const T* inB,
  T* out,
  int chwA,
  int chwB,
  int numBlocksA,
  int numBlocksB,
  int n
) {
  if(blockIdx.x < numBlocksA) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < chwA) {
      int nchwA = n*chwA;
      int chwOut = (chwA+chwB);

      int aIdx = index;
      int outIdx = index;
      while(aIdx < nchwA) {
        out[outIdx] = inA[aIdx];
        aIdx += chwA;
        outIdx += chwOut;
      }
    }
  }
  else {
    int index = (blockIdx.x - numBlocksA) * blockDim.x + threadIdx.x;
    if(index < chwB) {
      int nchwB = n*chwB;
      int chwOut = (chwA+chwB);

      int bIdx = index;
      int outIdx = chwA+index;
      while(bIdx < nchwB) {
        out[outIdx] = inB[bIdx];
        bIdx += chwB;
        outIdx += chwOut;
      }
    }
  }
}

template <typename T>
void customCudaChannelConcatTemplate(const T* inA, const T* inB, T* out, int chwA, int chwB, int n) {
  int blockSize = targetNumThreads;
  int numBlocksA = (chwA + blockSize-1) / blockSize;
  int numBlocksB = (chwB + blockSize-1) / blockSize;
  int numBlocks = numBlocksA + numBlocksB;
  channelConcatKernel<<<numBlocks, blockSize>>>(inA,inB,out,chwA,chwB,numBlocksA,numBlocksB,n);
}
template void customCudaChannelConcatTemplate<float>(const float* inA, const float* inB, float* out, int chwA, int chwB, int n);
template void customCudaChannelConcatTemplate<half>(const half* inA, const half* inB, half* out, int chwA, int chwB, int n);

void customCudaChannelConcat(const float* inA, const float* inB, float* out, int chwA, int chwB, int n) {
  customCudaChannelConcatTemplate<float>(inA,inB,out,chwA,chwB,n);
}
void customCudaChannelConcat(const half* inA, const half* inB, half* out, int chwA, int chwB, int n) {
  customCudaChannelConcatTemplate<half>(inA,inB,out,chwA,chwB,n);
}

//--------------------------------------------------------------------------------------------------------------

template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
  T c;
  __host__ __device__ linear_index_to_row_index(T c) : c(c) {}
  __host__ __device__ T operator()(T i) { return i / c; }
};

void customCudaPoolRowsSum(float* in, float* out, int n, int c) {

  thrust::device_ptr<float> inThrust = thrust::device_pointer_cast(in);
  thrust::device_ptr<float> outThrust = thrust::device_pointer_cast(out);

  thrust::reduce_by_key(
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(c)),
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(c)) + (n*c),
    inThrust,
    thrust::make_discard_iterator(),
    outThrust
  );

}

void customCudaPoolRowsMax(float* in, float* out, int n, int c) {

  thrust::device_ptr<float> inThrust = thrust::device_pointer_cast(in);
  thrust::device_ptr<float> outThrust = thrust::device_pointer_cast(out);

  thrust::reduce_by_key(
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(c)),
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(c)) + (n*c),
    inThrust,
    thrust::make_discard_iterator(),
    outThrust,
    thrust::equal_to<int>(),
    thrust::maximum<float>()
  );

}

//--------------------------------------------------------------------------------------------------------------

__global__
void nchwTransposeKernel(const float *in, float* out, int xSize, int ySize, int tileDim, int tileStride, int xySize)
{
  //+1 avoids bank conflicts
  extern __shared__ float tileNCHW[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int nc = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = xIdx + xSize * (yIdx+j) + xySize * nc;
      tileNCHW[(threadIdx.y+j)*tileDimP1 + threadIdx.x] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = outXIdx + ySize * (outYIdx+j) + xySize * nc;
      out[outIdx] = tileNCHW[threadIdx.x*tileDimP1 + threadIdx.y+j];
    }
  }
}

__global__
void nhwcTransposeKernel(const float *in, float* out, int xSize, int ySize, int cSize, int tileDim, int tileStride, int xycSize)
{
  //+1 reduces bank conflicts
  extern __shared__ float tileNHWC[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int cIdx = threadIdx.z;
  int n = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = cIdx + cSize * (xIdx + xSize * (yIdx+j)) + xycSize * n;
      tileNHWC[cIdx + cSize * ((threadIdx.y+j)*tileDimP1 + threadIdx.x)] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = cIdx + cSize * (outXIdx + ySize * (outYIdx+j)) + xycSize * n;
      out[outIdx] = tileNHWC[cIdx + cSize * (threadIdx.x*tileDimP1 + threadIdx.y+j)];
    }
  }
}

__global__
void nchwTransposeHalfKernel(const half *in, half* out, int xSize, int ySize, int tileDim, int tileStride, int xySize)
{
  //+1 avoids bank conflicts
  extern __shared__ half tileNCHWHALF[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int nc = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = xIdx + xSize * (yIdx+j) + xySize * nc;
      tileNCHWHALF[(threadIdx.y+j)*tileDimP1 + threadIdx.x] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = outXIdx + ySize * (outYIdx+j) + xySize * nc;
      out[outIdx] = tileNCHWHALF[threadIdx.x*tileDimP1 + threadIdx.y+j];
    }
  }
}

__global__
void nhwcTransposeHalfKernel(const half *in, half* out, int xSize, int ySize, int cSize, int tileDim, int tileStride, int xycSize)
{
  //+1 reduces bank conflicts
  extern __shared__ half tileNHWCHALF[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int cIdx = threadIdx.z;
  int n = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = cIdx + cSize * (xIdx + xSize * (yIdx+j)) + xycSize * n;
      tileNHWCHALF[cIdx + cSize * ((threadIdx.y+j)*tileDimP1 + threadIdx.x)] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = cIdx + cSize * (outXIdx + ySize * (outYIdx+j)) + xycSize * n;
      out[outIdx] = tileNHWCHALF[cIdx + cSize * (threadIdx.x*tileDimP1 + threadIdx.y+j)];
    }
  }
}

static void sharedNCHWTranspose(const void *in, void* out, int xSize, int ySize, int ncSize, bool isHalf) {
  if(ncSize > 65536)
    throw std::runtime_error("customCudaNCHWTranspose: ncSize too large");

  //TODO maybe tune these numbers, it varies by GPU
  //The first one should be the warp size, since it's set to what we need to avoid bank conflicts?
  //Or is it better to just make it xSize, to reduce overhead on top of 19x19?
  int tileDim = 32;
  int tileStride = targetNumThreads/tileDim;
  dim3 grid((xSize+tileDim-1)/tileDim,(ySize+tileDim-1)/tileDim,ncSize);
  dim3 threads(tileDim,tileStride,1);
  if(isHalf) {
    int sharedMemSize = sizeof(half)*tileDim*(tileDim+1);
    nchwTransposeHalfKernel<<<grid,threads,sharedMemSize>>>((const half*)in,(half*)out,xSize,ySize,tileDim,tileStride,xSize*ySize);
  }
  else {
    int sharedMemSize = sizeof(float)*tileDim*(tileDim+1);
    nchwTransposeKernel<<<grid,threads,sharedMemSize>>>((const float*)in,(float*)out,xSize,ySize,tileDim,tileStride,xSize*ySize);
  }
}
void customCudaNCHWTranspose(const float *in, float* out, int xSize, int ySize, int ncSize) {
  sharedNCHWTranspose(in,out,xSize,ySize,ncSize,false);
}
void customCudaNCHWTranspose(const half *in, half* out, int xSize, int ySize, int ncSize) {
  sharedNCHWTranspose(in,out,xSize,ySize,ncSize,true);
}

void sharedNHWCTranspose(const void *in, void* out, int xSize, int ySize, int cSize, int nSize, bool isHalf) {
  if(cSize > 64)
    throw std::runtime_error("customCudaNHWCTranspose: cSize too large");

  int tileDim = 1;
  while(tileDim * 2 * cSize <= targetNumThreads)
    tileDim *= 2;

  int tileStride = 1;
  if(tileDim > 32) {
    tileStride = tileDim / 32;
    tileDim = 32;
  }
  dim3 grid((xSize+tileDim-1)/tileDim,(ySize+tileDim-1)/tileDim,nSize);
  dim3 threads(tileDim,tileStride,cSize);

  if(isHalf) {
    int sharedMemSize = sizeof(half)*tileDim*(tileDim+1)*cSize;
    nhwcTransposeHalfKernel<<<grid,threads,sharedMemSize>>>((const half*)in,(half*)out,xSize,ySize,cSize,tileDim,tileStride,xSize*ySize*cSize);
  }
  else {
    int sharedMemSize = sizeof(float)*tileDim*(tileDim+1)*cSize;
    nhwcTransposeKernel<<<grid,threads,sharedMemSize>>>((const float*)in,(float*)out,xSize,ySize,cSize,tileDim,tileStride,xSize*ySize*cSize);
  }
}
void customCudaNHWCTranspose(const float *in, float* out, int xSize, int ySize, int cSize, int nSize) {
  sharedNHWCTranspose(in,out,xSize,ySize,cSize,nSize,false);
}
void customCudaNHWCTranspose(const half *in, half* out, int xSize, int ySize, int cSize, int nSize) {
  sharedNHWCTranspose(in,out,xSize,ySize,cSize,nSize,true);
}

//--------------------------------------------------------------------------------------------------------------


template <typename T>
__global__
void mirrorKernel(const T *in, T* out, int mSize, int subSize)
{
  int subIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int mIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int batchIdx = blockIdx.z;
  if(subIdx < subSize && mIdx < mSize) {
    int inIdx = subIdx + subSize * (mIdx + mSize * batchIdx);
    int outIdx = subIdx + subSize * ((mSize-mIdx-1) + mSize * batchIdx);
    out[outIdx] = in[inIdx];
  }
}

template <typename T>
void customCudaMirrorTemplate(const T *in, T* out, int batchSize, int mSize, int subSize) {
  if(batchSize > 65536)
    throw std::runtime_error("customCudaMirror: batchSize too large");
  if(mSize > 65536)
    throw std::runtime_error("customCudaMirror: mSize too large");

  int subThreads;
  int subBlocks;
  int mThreads;
  int mBlocks;

  if(subSize > targetNumThreads) {
    subThreads = targetNumThreads/2;
    subBlocks = (subSize + subThreads - 1) / subThreads;
    mThreads = 1;
    mBlocks = mSize;
  }
  else if(subSize > targetNumThreads/2) {
    subThreads = subSize;
    subBlocks = 1;
    mThreads = 1;
    mBlocks = mSize;
  }
  else {
    subThreads = subSize;
    subBlocks = 1;
    mThreads = targetNumThreads / subSize;
    mBlocks = (mSize + mThreads - 1) / mThreads;
  }

  dim3 grid(subBlocks,mBlocks,batchSize);
  dim3 threads(subThreads,mThreads,1);
  mirrorKernel<<<grid,threads>>>(in,out,mSize,subSize);
}

template <typename T>
void customCudaMirrorNCHWTemplate(const T *in, T* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  if(mirrorY && mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*cSize,ySize*xSize,1);
  else if(mirrorY)
    customCudaMirrorTemplate(in,out,batchSize*cSize,ySize,xSize);
  else if(mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*cSize*ySize,xSize,1);
  else
    hipMemcpyAsync(out,in,sizeof(T)*batchSize*cSize*ySize*xSize,hipMemcpyDeviceToDevice);
}

template <typename T>
void customCudaMirrorNHWCTemplate(const T *in, T* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  if(mirrorY && mirrorX)
    customCudaMirrorTemplate(in,out,batchSize,ySize*xSize,cSize);
  else if(mirrorY)
    customCudaMirrorTemplate(in,out,batchSize,ySize,xSize*cSize);
  else if(mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*ySize,xSize,cSize);
  else
    hipMemcpyAsync(out,in,sizeof(T)*batchSize*ySize*xSize*cSize,hipMemcpyDeviceToDevice);
}

void customCudaMirror(const float *in, float* out, int batchSize, int mSize, int subSize) {
  customCudaMirrorTemplate<float>(in,out,batchSize,mSize,subSize);
}
void customCudaMirrorNCHW(const float *in, float* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNCHWTemplate<float>(in,out,batchSize,cSize,ySize,xSize,mirrorY,mirrorX);
}
void customCudaMirrorNHWC(const float *in, float* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNHWCTemplate<float>(in,out,batchSize,ySize,xSize,cSize,mirrorY,mirrorX);
}

void customCudaMirror(const half *in, half* out, int batchSize, int mSize, int subSize) {
  customCudaMirrorTemplate<half>(in,out,batchSize,mSize,subSize);
}
void customCudaMirrorNCHW(const half *in, half* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNCHWTemplate<half>(in,out,batchSize,cSize,ySize,xSize,mirrorY,mirrorX);
}
void customCudaMirrorNHWC(const half *in, half* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNHWCTemplate<half>(in,out,batchSize,ySize,xSize,cSize,mirrorY,mirrorX);
}


//--------------------------------------------------------------------------------------------------------------

__global__
void copyToHalfKernel(const float *in, half* out, int n)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n) {
    out[idx] = __float2half(in[idx]);
  }
}
__global__
void copyFromHalfKernel(const half *in, float* out, int n)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n) {
    out[idx] = __half2float(in[idx]);
  }
}

void customCudaCopyToHalf(const float* in, half* out, int n) {
  int blockSize = targetNumThreads;
  int numBlocks = (n+blockSize-1)/blockSize;
  copyToHalfKernel<<<numBlocks, blockSize>>>(in,out,n);
}
void customCudaCopyFromHalf(const half* in, float* out, int n) {
  int blockSize = targetNumThreads;
  int numBlocks = (n+blockSize-1)/blockSize;
  copyFromHalfKernel<<<numBlocks, blockSize>>>(in,out,n);
}

//--------------------------------------------------------------------------------------------------------------


#ifdef CUDA_SUPPORTS_FP16
__global__
void addTensorInplaceHalfKernel(half *buf, const half* biases, int nSize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < nSize) {
    buf[idx] = __hadd(buf[idx],biases[idx]);
  }
}
#else
__global__
void addTensorInplaceHalfKernel(half *buf, const half* biases, int nSize)
{
  //Do nothing, FP16 not supported
}
#endif
void customCudaAddTensorInplace(half* buf, const half* biases, int nSize) {
  int blockSize = targetNumThreads;
  int numBlocks = (nSize+blockSize-1)/blockSize;
  addTensorInplaceHalfKernel<<<numBlocks, blockSize>>>(buf,biases,nSize);
}

//--------------------------------------------------------------------------------------------------------------


__global__
void addCBiasInplaceNCKernel(float *buf, const float* biases, int nSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int nIdx = blockIdx.y * blockDim.y + threadIdx.y;
  if(cIdx < cSize && nIdx < nSize) {
    int idx = nIdx * cSize + cIdx;
    buf[idx] = buf[idx] + biases[cIdx];
  }
}
#ifdef CUDA_SUPPORTS_FP16
__global__
void addCBiasInplaceNCHalfKernel(half *buf, const half* biases, int nSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int nIdx = blockIdx.y * blockDim.y + threadIdx.y;
  if(cIdx < cSize && nIdx < nSize) {
    int idx = nIdx * cSize + cIdx;
    buf[idx] = __hadd(buf[idx],biases[cIdx]);
  }
}
#else
__global__
void addCBiasInplaceNCHalfKernel(half *buf, const half* biases, int nSize, int cSize)
{
  //Do nothing, FP16 not supported
}
#endif

void sharedAddCBiasInplaceNC(void* buf, const void* biases, int nSize, int cSize, bool isHalf) {
  int cThreads;
  int cBlocks;
  int nThreads;
  int nBlocks;

  if(cSize > targetNumThreads) {
    cThreads = targetNumThreads/2;
    cBlocks = (cSize + cThreads - 1) / cThreads;
    nThreads = 1;
    nBlocks = nSize;
  }
  else if(cSize > targetNumThreads/2) {
    cThreads = cSize;
    cBlocks = 1;
    nThreads = 1;
    nBlocks = nSize;
  }
  else {
    cThreads = cSize;
    cBlocks = 1;
    nThreads = targetNumThreads / cSize;
    nBlocks = (nSize + nThreads - 1) / nThreads;
  }

  if(nBlocks > 65536)
    throw std::runtime_error("customCudaAddCBiasInplaceNC: nSize too large given cSize");

  dim3 grid(cBlocks,nBlocks,1);
  dim3 threads(cThreads,nThreads,1);

  if(isHalf)
    addCBiasInplaceNCHalfKernel<<<grid,threads>>>((half*)buf,(const half*)biases,nSize,cSize);
  else
    addCBiasInplaceNCKernel<<<grid,threads>>>((float*)buf,(const float*)biases,nSize,cSize);
}

void customCudaAddCBiasInplaceNC(float* buf, const float* biases, int nSize, int cSize) {
  sharedAddCBiasInplaceNC(buf,biases,nSize,cSize,false);
}
void customCudaAddCBiasInplaceNC(half* buf, const half* biases, int nSize, int cSize) {
  sharedAddCBiasInplaceNC(buf,biases,nSize,cSize,true);
}

//--------------------------------------------------------------------------------------------------------------

__global__
void addNCBiasInplaceNCHWKernel(float *buf, const float* biases, int cSize, int sSize)
{
  int sIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int cIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int ncIdx = nIdx * cSize + cIdx;
    int idx = ncIdx * sSize + sIdx;
    buf[idx] = buf[idx] + biases[ncIdx];
  }
}
#ifdef CUDA_SUPPORTS_FP16
__global__
void addNCBiasInplaceNCHWHalfKernel(half *buf, const half* biases, int cSize, int sSize)
{
  int sIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int cIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int ncIdx = nIdx * cSize + cIdx;
    int idx = ncIdx * sSize + sIdx;
    buf[idx] = __hadd(buf[idx],biases[ncIdx]);
  }
}
#else
__global__
void addNCBiasInplaceNCHWHalfKernel(half *buf, const half* biases, int cSize, int sSize) {
  //Do nothing, FP16 not supported
}
#endif

void sharedAddNCBiasInplaceNCHW(void *buf, const void* biases, int nSize, int cSize, int xySize, bool isHalf) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaAddNCBiasInplaceNCHW: nSize too large");
  if(cSize > 65536)
    throw std::runtime_error("customCudaAddNCBiasInplaceNCHW: cSize too large");

  int sSize = xySize;
  int sThreads;
  int sBlocks;
  int cThreads;
  int cBlocks;

  if(sSize > targetNumThreads) {
    sThreads = targetNumThreads/2;
    sBlocks = (sSize + sThreads - 1) / sThreads;
    cThreads = 1;
    cBlocks = cSize;
  }
  else if(sSize > targetNumThreads/2) {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = 1;
    cBlocks = cSize;
  }
  else {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = targetNumThreads / sSize;
    cBlocks = (cSize + cThreads - 1) / cThreads;
  }

  dim3 grid(sBlocks,cBlocks,nSize);
  dim3 threads(sThreads,cThreads,1);
  if(isHalf)
    addNCBiasInplaceNCHWHalfKernel<<<grid,threads>>>((half*)buf,(const half*)biases,cSize,sSize);
  else
    addNCBiasInplaceNCHWKernel<<<grid,threads>>>((float*)buf,(const float*)biases,cSize,sSize);
}

void customCudaAddNCBiasInplaceNCHW(float *buf, const float* biases, int nSize, int cSize, int xySize) {
  sharedAddNCBiasInplaceNCHW(buf,biases,nSize,cSize,xySize,false);
}
void customCudaAddNCBiasInplaceNCHW(half *buf, const half* biases, int nSize, int cSize, int xySize) {
  sharedAddNCBiasInplaceNCHW(buf,biases,nSize,cSize,xySize,true);
}

//--------------------------------------------------------------------------------------------------------------

#ifdef CUDA_SUPPORTS_FP16
__global__
void applyCScaleBiasNCHWHalfKernel(const half *in, half* out, const half* scale, const half* biases, int cSize, int sSize)
{
  int sIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int cIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int idx = (nIdx * cSize + cIdx) * sSize + sIdx;
    out[idx] = __hfma(in[idx],scale[cIdx],biases[cIdx]);
  }
}
#else
__global__
void applyCScaleBiasNCHWHalfKernel(const half *in, half* out, const half* scale, const half* biases, int cSize, int sSize)
{
  //Do nothing, FP16 not supported
}
#endif

void customCudaApplyCScaleBiasNCHW(const half* in, half* out, const half* scale, const half* biases, int nSize, int cSize, int xySize) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaApplyCScaleBiasNCHW: nSize too large");
  if(cSize > 65536)
    throw std::runtime_error("customCudaApplyCScaleBiasNCHW: cSize too large");

  int sSize = xySize;
  int sThreads;
  int sBlocks;
  int cThreads;
  int cBlocks;

  if(sSize > targetNumThreads) {
    sThreads = targetNumThreads/2;
    sBlocks = (sSize + sThreads - 1) / sThreads;
    cThreads = 1;
    cBlocks = cSize;
  }
  else if(sSize > targetNumThreads/2) {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = 1;
    cBlocks = cSize;
  }
  else {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = targetNumThreads / sSize;
    cBlocks = (cSize + cThreads - 1) / cThreads;
  }

  dim3 grid(sBlocks,cBlocks,nSize);
  dim3 threads(sThreads,cThreads,1);
  applyCScaleBiasNCHWHalfKernel<<<grid,threads>>>(in,out,scale,biases,cSize,sSize);
}
