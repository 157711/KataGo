#include "hip/hip_runtime.h"

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include <hip/hip_fp16.h>
#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/iterator/discard_iterator.h>

#include <stdexcept>

#include "../neuralnet/cudahelpers.h"

#if __CUDA_ARCH__ >= 530
#define CUDA_SUPPORTS_FP16
#endif

//TODO maybe tune this number, it varies by GPU
static const int targetNumThreads = 512;

//--------------------------------------------------------------------------------------------------------------

template <typename T>
__global__
void channelConcatKernel(
  const T* inA,
  const T* inB,
  T* out,
  int chwA,
  int chwB,
  int numBlocksA,
  int numBlocksB,
  int n
) {
  if(blockIdx.x < numBlocksA) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < chwA) {
      int nchwA = n*chwA;
      int chwOut = (chwA+chwB);

      int aIdx = index;
      int outIdx = index;
      while(aIdx < nchwA) {
        out[outIdx] = inA[aIdx];
        aIdx += chwA;
        outIdx += chwOut;
      }
    }
  }
  else {
    int index = (blockIdx.x - numBlocksA) * blockDim.x + threadIdx.x;
    if(index < chwB) {
      int nchwB = n*chwB;
      int chwOut = (chwA+chwB);

      int bIdx = index;
      int outIdx = chwA+index;
      while(bIdx < nchwB) {
        out[outIdx] = inB[bIdx];
        bIdx += chwB;
        outIdx += chwOut;
      }
    }
  }
}

template <typename T>
void customCudaChannelConcatTemplate(const T* inA, const T* inB, T* out, int chwA, int chwB, int n) {
  int blockSize = targetNumThreads;
  int numBlocksA = (chwA + blockSize-1) / blockSize;
  int numBlocksB = (chwB + blockSize-1) / blockSize;
  int numBlocks = numBlocksA + numBlocksB;
  channelConcatKernel<<<numBlocks, blockSize>>>(inA,inB,out,chwA,chwB,numBlocksA,numBlocksB,n);
}
template void customCudaChannelConcatTemplate<float>(const float* inA, const float* inB, float* out, int chwA, int chwB, int n);
template void customCudaChannelConcatTemplate<half>(const half* inA, const half* inB, half* out, int chwA, int chwB, int n);

void customCudaChannelConcat(const float* inA, const float* inB, float* out, int chwA, int chwB, int n) {
  customCudaChannelConcatTemplate<float>(inA,inB,out,chwA,chwB,n);
}
void customCudaChannelConcat(const half* inA, const half* inB, half* out, int chwA, int chwB, int n) {
  customCudaChannelConcatTemplate<half>(inA,inB,out,chwA,chwB,n);
}

//--------------------------------------------------------------------------------------------------------------

template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
  T len;
  __host__ __device__ linear_index_to_row_index(T len) : len(len) {}
  __host__ __device__ T operator()(T i) { return i / len; }
};

void customCudaPoolRowsSumNCHW(float* in, float* out, int nc, int xy) {
  thrust::device_ptr<float> inThrust = thrust::device_pointer_cast(in);
  thrust::device_ptr<float> outThrust = thrust::device_pointer_cast(out);

  thrust::reduce_by_key(
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(xy)),
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(xy)) + (nc*xy),
    inThrust,
    thrust::make_discard_iterator(),
    outThrust
  );
}

void customCudaPoolRowsMaxNCHW(float* in, float* out, int nc, int xy) {
  thrust::device_ptr<float> inThrust = thrust::device_pointer_cast(in);
  thrust::device_ptr<float> outThrust = thrust::device_pointer_cast(out);

  thrust::reduce_by_key(
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(xy)),
    thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(xy)) + (nc*xy),
    inThrust,
    thrust::make_discard_iterator(),
    outThrust,
    thrust::equal_to<int>(),
    thrust::maximum<float>()
  );
}

__global__
void sumChannelsNHWCKernel(const float *in, float* out, int xySize, int cSize)
{
  extern __shared__ float sumPoolNHWCShared[];
  int cId = threadIdx.x;
  int cBlockDim = blockDim.x;
  int xyId = threadIdx.y;
  int xyBlockDim = blockDim.y;

  int cIdx = blockIdx.x * cBlockDim + cId;
  int nIdx = blockIdx.z;
  int sharedIdx = cId + cBlockDim * xyId;
  int xycSize = xySize*cSize;

  sumPoolNHWCShared[sharedIdx] = 0;

  if(cIdx < cSize) {
    int xyIdx = xyId;
    while(xyIdx < xySize) {
      sumPoolNHWCShared[sharedIdx] += in[cIdx + xyIdx * cSize + nIdx * xycSize];
      xyIdx += xyBlockDim;
    }
  }
  __syncthreads();

  for(int s = xyBlockDim>>1; s > 0; s >>= 1) {
    if(xyId < s) {
      sumPoolNHWCShared[sharedIdx] += sumPoolNHWCShared[sharedIdx + cBlockDim * s];
    }
    __syncthreads();
  }
  if(xyId == 0 && cIdx < cSize)
    out[cIdx + nIdx * cSize] = sumPoolNHWCShared[sharedIdx];
}
__global__
void maxChannelsNHWCKernel(const float *in, float* out, int xySize, int cSize)
{
  extern __shared__ float maxPoolNHWCShared[];
  int cId = threadIdx.x;
  int cBlockDim = blockDim.x;
  int xyId = threadIdx.y;
  int xyBlockDim = blockDim.y;

  int cIdx = blockIdx.x * cBlockDim + cId;
  int nIdx = blockIdx.z;
  int sharedIdx = cId + cBlockDim * xyId;
  int xycSize = xySize*cSize;

  maxPoolNHWCShared[sharedIdx] = 0;

  if(cIdx < cSize) {
    int xyIdx = xyId;
    while(xyIdx < xySize) {
      maxPoolNHWCShared[sharedIdx] = fmaxf(maxPoolNHWCShared[sharedIdx],in[cIdx + xyIdx * cSize + nIdx * xycSize]);
      xyIdx += xyBlockDim;
    }
  }
  __syncthreads();

  for(int s = xyBlockDim>>1; s > 0; s >>= 1) {
    if(xyId < s) {
      maxPoolNHWCShared[sharedIdx] = fmaxf(maxPoolNHWCShared[sharedIdx],maxPoolNHWCShared[sharedIdx + cBlockDim * s]);
    }
    __syncthreads();
  }
  if(xyId == 0 && cIdx < cSize)
    out[cIdx + nIdx * cSize] = maxPoolNHWCShared[sharedIdx];
}

void customCudaPoolRowsSumNHWC(const float* in, float* out, int nSize, int xySize, int cSize) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaPoolRowsSumNHWC: nSize too large");

  //Use up to two warps worth of threads along the channel dimension, which is the
  //most compact
  int cThreads = 1;
  while(cThreads < 64 && cThreads < cSize/2)
    cThreads *= 2;
  int cBlocks = (cSize + cThreads - 1) / cThreads;

  //Distribute the extra threads to perform parallel reduction along the xy dimension.
  int xyThreads = (targetNumThreads < cThreads) ? 1 : (targetNumThreads / cThreads);

  //We need one shared memory spot per thread
  int sharedMemSize = sizeof(float) * cThreads * xyThreads;

  dim3 grid(cBlocks,1,nSize);
  dim3 threads(cThreads,xyThreads,1);
  sumChannelsNHWCKernel<<<grid,threads,sharedMemSize>>>(in,out,xySize,cSize);
}

void customCudaPoolRowsMaxNHWC(const float* in, float* out, int nSize, int xySize, int cSize) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaPoolRowsMaxNHWC: nSize too large");

  //Use up to two warps worth of threads along the channel dimension, which is the
  //most compact
  int cThreads = 1;
  while(cThreads < 64 && cThreads < cSize/2)
    cThreads *= 2;
  int cBlocks = (cSize + cThreads - 1) / cThreads;

  //Distribute the extra threads to perform parallel reduction along the xy dimension.
  int xyThreads = (targetNumThreads < cThreads) ? 1 : (targetNumThreads / cThreads);

  //We need one shared memory spot per thread
  int sharedMemSize = sizeof(float) * cThreads * xyThreads;

  dim3 grid(cBlocks,1,nSize);
  dim3 threads(cThreads,xyThreads,1);
  maxChannelsNHWCKernel<<<grid,threads,sharedMemSize>>>(in,out,xySize,cSize);
}


//--------------------------------------------------------------------------------------------------------------

__global__
void nchwTransposeKernel(const float *in, float* out, int xSize, int ySize, int tileDim, int tileStride, int xySize)
{
  //+1 avoids bank conflicts
  extern __shared__ float tileNCHW[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int nc = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = xIdx + xSize * (yIdx+j) + xySize * nc;
      tileNCHW[(threadIdx.y+j)*tileDimP1 + threadIdx.x] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = outXIdx + ySize * (outYIdx+j) + xySize * nc;
      out[outIdx] = tileNCHW[threadIdx.x*tileDimP1 + threadIdx.y+j];
    }
  }
}

__global__
void nhwcTransposeKernel(const float *in, float* out, int xSize, int ySize, int cSize, int tileDim, int tileStride, int xycSize)
{
  //+1 reduces bank conflicts
  extern __shared__ float tileNHWC[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int cIdx = threadIdx.z;
  int n = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = cIdx + cSize * (xIdx + xSize * (yIdx+j)) + xycSize * n;
      tileNHWC[cIdx + cSize * ((threadIdx.y+j)*tileDimP1 + threadIdx.x)] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = cIdx + cSize * (outXIdx + ySize * (outYIdx+j)) + xycSize * n;
      out[outIdx] = tileNHWC[cIdx + cSize * (threadIdx.x*tileDimP1 + threadIdx.y+j)];
    }
  }
}

__global__
void nchwTransposeHalfKernel(const half *in, half* out, int xSize, int ySize, int tileDim, int tileStride, int xySize)
{
  //+1 avoids bank conflicts
  extern __shared__ half tileNCHWHALF[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int nc = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = xIdx + xSize * (yIdx+j) + xySize * nc;
      tileNCHWHALF[(threadIdx.y+j)*tileDimP1 + threadIdx.x] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = outXIdx + ySize * (outYIdx+j) + xySize * nc;
      out[outIdx] = tileNCHWHALF[threadIdx.x*tileDimP1 + threadIdx.y+j];
    }
  }
}

__global__
void nhwcTransposeHalfKernel(const half *in, half* out, int xSize, int ySize, int cSize, int tileDim, int tileStride, int xycSize)
{
  //+1 reduces bank conflicts
  extern __shared__ half tileNHWCHALF[];
  int tileDimP1 = tileDim+1;

  int xIdx = blockIdx.x * tileDim + threadIdx.x;
  int yIdx = blockIdx.y * tileDim + threadIdx.y;
  int cIdx = threadIdx.z;
  int n = blockIdx.z;
  if(xIdx < xSize) {
    for(int j = 0; j < tileDim && yIdx+j < ySize; j += tileStride) {
      int inIdx = cIdx + cSize * (xIdx + xSize * (yIdx+j)) + xycSize * n;
      tileNHWCHALF[cIdx + cSize * ((threadIdx.y+j)*tileDimP1 + threadIdx.x)] = in[inIdx];
    }
  }

  __syncthreads();

  //Transpose idx
  int outXIdx = blockIdx.y * tileDim + threadIdx.x;
  int outYIdx = blockIdx.x * tileDim + threadIdx.y;

  if(outXIdx < ySize) {
    for(int j = 0; j < tileDim && outYIdx+j < xSize; j += tileStride) {
      int outIdx = cIdx + cSize * (outXIdx + ySize * (outYIdx+j)) + xycSize * n;
      out[outIdx] = tileNHWCHALF[cIdx + cSize * (threadIdx.x*tileDimP1 + threadIdx.y+j)];
    }
  }
}

static void sharedNCHWTranspose(const void *in, void* out, int xSize, int ySize, int ncSize, bool isHalf) {
  if(ncSize > 65536)
    throw std::runtime_error("customCudaNCHWTranspose: ncSize too large");

  //TODO maybe tune these numbers, it varies by GPU
  //The first one should be the warp size, since it's set to what we need to avoid bank conflicts?
  //Or is it better to just make it xSize, to reduce overhead on top of 19x19?
  int tileDim = 32;
  int tileStride = targetNumThreads/tileDim;
  dim3 grid((xSize+tileDim-1)/tileDim,(ySize+tileDim-1)/tileDim,ncSize);
  dim3 threads(tileDim,tileStride,1);
  if(isHalf) {
    int sharedMemSize = sizeof(half)*tileDim*(tileDim+1);
    nchwTransposeHalfKernel<<<grid,threads,sharedMemSize>>>((const half*)in,(half*)out,xSize,ySize,tileDim,tileStride,xSize*ySize);
  }
  else {
    int sharedMemSize = sizeof(float)*tileDim*(tileDim+1);
    nchwTransposeKernel<<<grid,threads,sharedMemSize>>>((const float*)in,(float*)out,xSize,ySize,tileDim,tileStride,xSize*ySize);
  }
}
void customCudaNCHWTranspose(const float *in, float* out, int xSize, int ySize, int ncSize) {
  sharedNCHWTranspose(in,out,xSize,ySize,ncSize,false);
}
void customCudaNCHWTranspose(const half *in, half* out, int xSize, int ySize, int ncSize) {
  sharedNCHWTranspose(in,out,xSize,ySize,ncSize,true);
}

void sharedNHWCTranspose(const void *in, void* out, int xSize, int ySize, int cSize, int nSize, bool isHalf) {
  if(cSize > 64)
    throw std::runtime_error("customCudaNHWCTranspose: cSize too large");

  int tileDim = 1;
  while(tileDim * 2 * cSize <= targetNumThreads)
    tileDim *= 2;

  int tileStride = 1;
  if(tileDim > 32) {
    tileStride = tileDim / 32;
    tileDim = 32;
  }
  dim3 grid((xSize+tileDim-1)/tileDim,(ySize+tileDim-1)/tileDim,nSize);
  dim3 threads(tileDim,tileStride,cSize);

  if(isHalf) {
    int sharedMemSize = sizeof(half)*tileDim*(tileDim+1)*cSize;
    nhwcTransposeHalfKernel<<<grid,threads,sharedMemSize>>>((const half*)in,(half*)out,xSize,ySize,cSize,tileDim,tileStride,xSize*ySize*cSize);
  }
  else {
    int sharedMemSize = sizeof(float)*tileDim*(tileDim+1)*cSize;
    nhwcTransposeKernel<<<grid,threads,sharedMemSize>>>((const float*)in,(float*)out,xSize,ySize,cSize,tileDim,tileStride,xSize*ySize*cSize);
  }
}
void customCudaNHWCTranspose(const float *in, float* out, int xSize, int ySize, int cSize, int nSize) {
  sharedNHWCTranspose(in,out,xSize,ySize,cSize,nSize,false);
}
void customCudaNHWCTranspose(const half *in, half* out, int xSize, int ySize, int cSize, int nSize) {
  sharedNHWCTranspose(in,out,xSize,ySize,cSize,nSize,true);
}

//--------------------------------------------------------------------------------------------------------------


template <typename T>
__global__
void mirrorKernel(const T *in, T* out, int mSize, int subSize)
{
  int subIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int mIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int batchIdx = blockIdx.z;
  if(subIdx < subSize && mIdx < mSize) {
    int inIdx = subIdx + subSize * (mIdx + mSize * batchIdx);
    int outIdx = subIdx + subSize * ((mSize-mIdx-1) + mSize * batchIdx);
    out[outIdx] = in[inIdx];
  }
}

template <typename T>
void customCudaMirrorTemplate(const T *in, T* out, int batchSize, int mSize, int subSize) {
  if(batchSize > 65536)
    throw std::runtime_error("customCudaMirror: batchSize too large");
  if(mSize > 65536)
    throw std::runtime_error("customCudaMirror: mSize too large");

  int subThreads;
  int subBlocks;
  int mThreads;
  int mBlocks;

  if(subSize > targetNumThreads) {
    subThreads = targetNumThreads/2;
    subBlocks = (subSize + subThreads - 1) / subThreads;
    mThreads = 1;
    mBlocks = mSize;
  }
  else if(subSize > targetNumThreads/2) {
    subThreads = subSize;
    subBlocks = 1;
    mThreads = 1;
    mBlocks = mSize;
  }
  else {
    subThreads = subSize;
    subBlocks = 1;
    mThreads = targetNumThreads / subSize;
    mBlocks = (mSize + mThreads - 1) / mThreads;
  }

  dim3 grid(subBlocks,mBlocks,batchSize);
  dim3 threads(subThreads,mThreads,1);
  mirrorKernel<<<grid,threads>>>(in,out,mSize,subSize);
}

template <typename T>
void customCudaMirrorNCHWTemplate(const T *in, T* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  if(mirrorY && mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*cSize,ySize*xSize,1);
  else if(mirrorY)
    customCudaMirrorTemplate(in,out,batchSize*cSize,ySize,xSize);
  else if(mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*cSize*ySize,xSize,1);
  else
    hipMemcpyAsync(out,in,sizeof(T)*batchSize*cSize*ySize*xSize,hipMemcpyDeviceToDevice);
}

template <typename T>
void customCudaMirrorNHWCTemplate(const T *in, T* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  if(mirrorY && mirrorX)
    customCudaMirrorTemplate(in,out,batchSize,ySize*xSize,cSize);
  else if(mirrorY)
    customCudaMirrorTemplate(in,out,batchSize,ySize,xSize*cSize);
  else if(mirrorX)
    customCudaMirrorTemplate(in,out,batchSize*ySize,xSize,cSize);
  else
    hipMemcpyAsync(out,in,sizeof(T)*batchSize*ySize*xSize*cSize,hipMemcpyDeviceToDevice);
}

void customCudaMirror(const float *in, float* out, int batchSize, int mSize, int subSize) {
  customCudaMirrorTemplate<float>(in,out,batchSize,mSize,subSize);
}
void customCudaMirrorNCHW(const float *in, float* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNCHWTemplate<float>(in,out,batchSize,cSize,ySize,xSize,mirrorY,mirrorX);
}
void customCudaMirrorNHWC(const float *in, float* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNHWCTemplate<float>(in,out,batchSize,ySize,xSize,cSize,mirrorY,mirrorX);
}

void customCudaMirror(const half *in, half* out, int batchSize, int mSize, int subSize) {
  customCudaMirrorTemplate<half>(in,out,batchSize,mSize,subSize);
}
void customCudaMirrorNCHW(const half *in, half* out, int batchSize, int cSize, int ySize, int xSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNCHWTemplate<half>(in,out,batchSize,cSize,ySize,xSize,mirrorY,mirrorX);
}
void customCudaMirrorNHWC(const half *in, half* out, int batchSize, int ySize, int xSize, int cSize, bool mirrorY, bool mirrorX) {
  customCudaMirrorNHWCTemplate<half>(in,out,batchSize,ySize,xSize,cSize,mirrorY,mirrorX);
}


//--------------------------------------------------------------------------------------------------------------

__global__
void copyToHalfKernel(const float *in, half* out, int n)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n) {
    out[idx] = __float2half(in[idx]);
  }
}
__global__
void copyFromHalfKernel(const half *in, float* out, int n)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n) {
    out[idx] = __half2float(in[idx]);
  }
}

void customCudaCopyToHalf(const float* in, half* out, int n) {
  int blockSize = targetNumThreads;
  int numBlocks = (n+blockSize-1)/blockSize;
  copyToHalfKernel<<<numBlocks, blockSize>>>(in,out,n);
}
void customCudaCopyFromHalf(const half* in, float* out, int n) {
  int blockSize = targetNumThreads;
  int numBlocks = (n+blockSize-1)/blockSize;
  copyFromHalfKernel<<<numBlocks, blockSize>>>(in,out,n);
}

//--------------------------------------------------------------------------------------------------------------


#ifdef CUDA_SUPPORTS_FP16
__global__
void addTensorInplaceHalfKernel(half *buf, const half* biases, int nSize)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < nSize) {
    buf[idx] = __hadd(buf[idx],biases[idx]);
  }
}
#else
__global__
void addTensorInplaceHalfKernel(half *buf, const half* biases, int nSize)
{
  //Do nothing, FP16 not supported
}
#endif
void customCudaAddTensorInplace(half* buf, const half* biases, int nSize) {
  int blockSize = targetNumThreads;
  int numBlocks = (nSize+blockSize-1)/blockSize;
  addTensorInplaceHalfKernel<<<numBlocks, blockSize>>>(buf,biases,nSize);
}

//--------------------------------------------------------------------------------------------------------------


__global__
void addCBiasInplaceNCKernel(float *buf, const float* biases, int nSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int nIdx = blockIdx.y * blockDim.y + threadIdx.y;
  if(cIdx < cSize && nIdx < nSize) {
    int idx = nIdx * cSize + cIdx;
    buf[idx] = buf[idx] + biases[cIdx];
  }
}
#ifdef CUDA_SUPPORTS_FP16
__global__
void addCBiasInplaceNCHalfKernel(half *buf, const half* biases, int nSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int nIdx = blockIdx.y * blockDim.y + threadIdx.y;
  if(cIdx < cSize && nIdx < nSize) {
    int idx = nIdx * cSize + cIdx;
    buf[idx] = __hadd(buf[idx],biases[cIdx]);
  }
}
#else
__global__
void addCBiasInplaceNCHalfKernel(half *buf, const half* biases, int nSize, int cSize)
{
  //Do nothing, FP16 not supported
}
#endif

void sharedAddCBiasInplaceNC(void* buf, const void* biases, int nSize, int cSize, bool isHalf) {
  int cThreads;
  int cBlocks;
  int nThreads;
  int nBlocks;

  if(cSize > targetNumThreads) {
    cThreads = targetNumThreads/2;
    cBlocks = (cSize + cThreads - 1) / cThreads;
    nThreads = 1;
    nBlocks = nSize;
  }
  else if(cSize > targetNumThreads/2) {
    cThreads = cSize;
    cBlocks = 1;
    nThreads = 1;
    nBlocks = nSize;
  }
  else {
    cThreads = cSize;
    cBlocks = 1;
    nThreads = targetNumThreads / cSize;
    nBlocks = (nSize + nThreads - 1) / nThreads;
  }

  if(nBlocks > 65536)
    throw std::runtime_error("customCudaAddCBiasInplaceNC: nSize too large given cSize");

  dim3 grid(cBlocks,nBlocks,1);
  dim3 threads(cThreads,nThreads,1);

  if(isHalf)
    addCBiasInplaceNCHalfKernel<<<grid,threads>>>((half*)buf,(const half*)biases,nSize,cSize);
  else
    addCBiasInplaceNCKernel<<<grid,threads>>>((float*)buf,(const float*)biases,nSize,cSize);
}

void customCudaAddCBiasInplaceNC(float* buf, const float* biases, int nSize, int cSize) {
  sharedAddCBiasInplaceNC(buf,biases,nSize,cSize,false);
}
void customCudaAddCBiasInplaceNC(half* buf, const half* biases, int nSize, int cSize) {
  sharedAddCBiasInplaceNC(buf,biases,nSize,cSize,true);
}

//--------------------------------------------------------------------------------------------------------------

__global__
void addNCBiasInplaceNCHWKernel(float *buf, const float* biases, int cSize, int sSize)
{
  int sIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int cIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int ncIdx = nIdx * cSize + cIdx;
    int idx = ncIdx * sSize + sIdx;
    buf[idx] = buf[idx] + biases[ncIdx];
  }
}
#ifdef CUDA_SUPPORTS_FP16
__global__
void addNCBiasInplaceNCHWHalfKernel(half *buf, const half* biases, int cSize, int sSize)
{
  int sIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int cIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int ncIdx = nIdx * cSize + cIdx;
    int idx = ncIdx * sSize + sIdx;
    buf[idx] = __hadd(buf[idx],biases[ncIdx]);
  }
}
#else
__global__
void addNCBiasInplaceNCHWHalfKernel(half *buf, const half* biases, int cSize, int sSize) {
  //Do nothing, FP16 not supported
}
#endif

void sharedAddNCBiasInplaceNCHW(void *buf, const void* biases, int nSize, int cSize, int xySize, bool isHalf) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaAddNCBiasInplaceNCHW: nSize too large");
  if(cSize > 65536)
    throw std::runtime_error("customCudaAddNCBiasInplaceNCHW: cSize too large");

  int sSize = xySize;
  int sThreads;
  int sBlocks;
  int cThreads;
  int cBlocks;

  if(sSize > targetNumThreads) {
    sThreads = targetNumThreads/2;
    sBlocks = (sSize + sThreads - 1) / sThreads;
    cThreads = 1;
    cBlocks = cSize;
  }
  else if(sSize > targetNumThreads/2) {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = 1;
    cBlocks = cSize;
  }
  else {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = targetNumThreads / sSize;
    cBlocks = (cSize + cThreads - 1) / cThreads;
  }

  dim3 grid(sBlocks,cBlocks,nSize);
  dim3 threads(sThreads,cThreads,1);
  if(isHalf)
    addNCBiasInplaceNCHWHalfKernel<<<grid,threads>>>((half*)buf,(const half*)biases,cSize,sSize);
  else
    addNCBiasInplaceNCHWKernel<<<grid,threads>>>((float*)buf,(const float*)biases,cSize,sSize);
}

void customCudaAddNCBiasInplaceNCHW(float *buf, const float* biases, int nSize, int cSize, int xySize) {
  sharedAddNCBiasInplaceNCHW(buf,biases,nSize,cSize,xySize,false);
}
void customCudaAddNCBiasInplaceNCHW(half *buf, const half* biases, int nSize, int cSize, int xySize) {
  sharedAddNCBiasInplaceNCHW(buf,biases,nSize,cSize,xySize,true);
}

//--------------------------------------------------------------------------------------------------------------

__global__
void addNCBiasInplaceNHWCKernel(float *buf, const float* biases, int sSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int sIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int ncIdx = nIdx * cSize + cIdx;
    int idx = (nIdx * sSize + sIdx) * cSize + cIdx;
    buf[idx] = buf[idx] + biases[ncIdx];
  }
}
#ifdef CUDA_SUPPORTS_FP16
__global__
void addNCBiasInplaceNHWCHalfKernel(half *buf, const half* biases, int sSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int sIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int ncIdx = nIdx * cSize + cIdx;
    int idx = (nIdx * sSize + sIdx) * cSize + cIdx;
    buf[idx] = __hadd(buf[idx],biases[ncIdx]);
  }
}
#else
__global__
void addNCBiasInplaceNHWCHalfKernel(half *buf, const half* biases, int sSize, int cSize)
{
  //Do nothing, FP16 not supported
}
#endif

void sharedAddNCBiasInplaceNHWC(void *buf, const void* biases, int nSize, int xySize, int cSize, bool isHalf) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaAddNCBiasInplaceNHWC: nSize too large");
  if(xySize > 65536)
    throw std::runtime_error("customCudaAddNCBiasInplaceNHWC: xySize too large");

  int sSize = xySize;
  int cThreads;
  int cBlocks;
  int sThreads;
  int sBlocks;

  if(cSize > targetNumThreads) {
    cThreads = targetNumThreads/2;
    cBlocks = (cSize + cThreads - 1) / cThreads;
    sThreads = 1;
    sBlocks = sSize;
  }
  else if(cSize > targetNumThreads/2) {
    cThreads = cSize;
    cBlocks = 1;
    sThreads = 1;
    sBlocks = sSize;
  }
  else {
    cThreads = cSize;
    cBlocks = 1;
    sThreads = targetNumThreads / cSize;
    sBlocks = (sSize + sThreads - 1) / sThreads;
  }

  dim3 grid(cBlocks,sBlocks,nSize);
  dim3 threads(cThreads,sThreads,1);
  if(isHalf)
    addNCBiasInplaceNHWCHalfKernel<<<grid,threads>>>((half*)buf,(const half*)biases,sSize,cSize);
  else
    addNCBiasInplaceNHWCKernel<<<grid,threads>>>((float*)buf,(const float*)biases,sSize,cSize);
}

void customCudaAddNCBiasInplaceNHWC(float *buf, const float* biases, int nSize, int xySize, int cSize) {
  sharedAddNCBiasInplaceNHWC(buf,biases,nSize,xySize,cSize,false);
}
void customCudaAddNCBiasInplaceNHWC(half *buf, const half* biases, int nSize, int xySize, int cSize) {
  sharedAddNCBiasInplaceNHWC(buf,biases,nSize,xySize,cSize,true);
}

//--------------------------------------------------------------------------------------------------------------

__global__
void applyCScaleBiasNCHWKernel(const float *in, float* out, const float* scale, const float* biases, int cSize, int sSize)
{
  int sIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int cIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int idx = (nIdx * cSize + cIdx) * sSize + sIdx;
    out[idx] = in[idx] * scale[cIdx] + biases[cIdx];
  }
}
#ifdef CUDA_SUPPORTS_FP16
__global__
void applyCScaleBiasNCHWHalfKernel(const half *in, half* out, const half* scale, const half* biases, int cSize, int sSize)
{
  int sIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int cIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int idx = (nIdx * cSize + cIdx) * sSize + sIdx;
    out[idx] = __hfma(in[idx],scale[cIdx],biases[cIdx]);
  }
}
#else
__global__
void applyCScaleBiasNCHWHalfKernel(const half *in, half* out, const half* scale, const half* biases, int cSize, int sSize)
{
  //Do nothing, FP16 not supported
}
#endif

void sharedApplyCScaleBiasNCHW(const void* in, void* out, const void* scale, const void* biases, int nSize, int cSize, int xySize, bool isHalf) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaApplyCScaleBiasNCHW: nSize too large");
  if(cSize > 65536)
    throw std::runtime_error("customCudaApplyCScaleBiasNCHW: cSize too large");

  int sSize = xySize;
  int sThreads;
  int sBlocks;
  int cThreads;
  int cBlocks;

  if(sSize > targetNumThreads) {
    sThreads = targetNumThreads/2;
    sBlocks = (sSize + sThreads - 1) / sThreads;
    cThreads = 1;
    cBlocks = cSize;
  }
  else if(sSize > targetNumThreads/2) {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = 1;
    cBlocks = cSize;
  }
  else {
    sThreads = sSize;
    sBlocks = 1;
    cThreads = targetNumThreads / sSize;
    cBlocks = (cSize + cThreads - 1) / cThreads;
  }

  dim3 grid(sBlocks,cBlocks,nSize);
  dim3 threads(sThreads,cThreads,1);
  if(isHalf)
    applyCScaleBiasNCHWHalfKernel<<<grid,threads>>>((const half*)in,(half*)out,(const half*)scale,(const half*)biases,cSize,sSize);
  else
    applyCScaleBiasNCHWKernel<<<grid,threads>>>((const float*)in,(float*)out,(const float*)scale,(const float*)biases,cSize,sSize);
}

void customCudaApplyCScaleBiasNCHW(const float* in, float* out, const float* scale, const float* biases, int nSize, int cSize, int xySize) {
  sharedApplyCScaleBiasNCHW(in,out,scale,biases,nSize,cSize,xySize,false);
}
void customCudaApplyCScaleBiasNCHW(const half* in, half* out, const half* scale, const half* biases, int nSize, int cSize, int xySize) {
  sharedApplyCScaleBiasNCHW(in,out,scale,biases,nSize,cSize,xySize,true);
}


//--------------------------------------------------------------------------------------------------------------

__global__
void applyCScaleBiasNHWCKernel(const float* in, float* out, const float* scale, const float* biases, int sSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int sIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int idx = (nIdx * sSize + sIdx) * cSize + cIdx;
    out[idx] = in[idx] * scale[cIdx] + biases[cIdx];
  }
}
#ifdef CUDA_SUPPORTS_FP16
__global__
void applyCScaleBiasNHWCHalfKernel(const half* in, half* out, const half* scale, const half* biases, int sSize, int cSize)
{
  int cIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int sIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nIdx = blockIdx.z;
  if(cIdx < cSize && sIdx < sSize) {
    int idx = (nIdx * sSize + sIdx) * cSize + cIdx;
    out[idx] = __hfma(in[idx],scale[cIdx],biases[cIdx]);
  }
}
#else
__global__
void applyCScaleBiasNHWCHalfKernel(const half* in, half* out, const half* scale, const half* biases, int sSize, int cSize)
{
  //Do nothing, FP16 not supported
}
#endif

void sharedApplyCScaleBiasNHWC(const void* in, void* out, const void* scale, const void* biases, int nSize, int xySize, int cSize, bool isHalf) {
  if(nSize > 65536)
    throw std::runtime_error("customCudaApplyCScaleBiasNHWC: nSize too large");
  if(xySize > 65536)
    throw std::runtime_error("customCudaApplyCScaleBiasNHWC: xySize too large");

  int sSize = xySize;
  int cThreads;
  int cBlocks;
  int sThreads;
  int sBlocks;

  if(cSize > targetNumThreads) {
    cThreads = targetNumThreads/2;
    cBlocks = (cSize + cThreads - 1) / cThreads;
    sThreads = 1;
    sBlocks = sSize;
  }
  else if(cSize > targetNumThreads/2) {
    cThreads = cSize;
    cBlocks = 1;
    sThreads = 1;
    sBlocks = sSize;
  }
  else {
    cThreads = cSize;
    cBlocks = 1;
    sThreads = targetNumThreads / cSize;
    sBlocks = (sSize + sThreads - 1) / sThreads;
  }

  dim3 grid(cBlocks,sBlocks,nSize);
  dim3 threads(cThreads,sThreads,1);
  if(isHalf)
    applyCScaleBiasNHWCHalfKernel<<<grid,threads>>>((const half*)in,(half*)out,(const half*)scale,(const half*)biases,sSize,cSize);
  else
    applyCScaleBiasNHWCKernel<<<grid,threads>>>((const float*)in,(float*)out,(const float*)scale,(const float*)biases,sSize,cSize);
}

void customCudaApplyCScaleBiasNHWC(const float* in, float* out, const float* scale, const float* biases, int nSize, int xySize, int cSize) {
  sharedApplyCScaleBiasNHWC(in,out,scale,biases,nSize,xySize,cSize,false);
}
void customCudaApplyCScaleBiasNHWC(const half* in, half* out, const half* scale, const half* biases, int nSize, int xySize, int cSize) {
  sharedApplyCScaleBiasNHWC(in,out,scale,biases,nSize,xySize,cSize,true);
}
